#include "hip/hip_runtime.h"
%%cu
#include <hip/hip_runtime.h>
#include <>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include<stdlib.h>

//using namespace std;

__global__ void add(int *a, int *b, int *c) 
{   
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];  
 } 

void charger(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = i;
    }

#define N 12  
int main(void) 
{   
    int *a, *b, *c;  // host (CPU) copies of a, b, c   
    int *d_a, *d_b, *d_c;  // device (GPU) copies of a, b, c   
    int size = N * sizeof(int);      
 
    // Alloc space for device copies of a, b, c   
    hipMalloc((void **)&d_a, size);   
    hipMalloc((void **)&d_b, size);   
    hipMalloc((void **)&d_c, size);    
 
    // Alloc space for host copies of a, b, c and setup input values   
    a = (int *)malloc(size); 
    charger(a, N);   
    b = (int *)malloc(size); 
    charger(b, N);   
    c = (int *)malloc(size);  
 
    // Copy inputs to device   
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);   
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);    
 
    // Launch add() kernel on GPU with N blocks   
    add<<<N,1>>>(d_a, d_b, d_c);    
    
    // Copy result back to host   
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);   
    printf("Le vecteur C = A + B :\n") ;
    for (int i=0;i<N;i++)
      printf("%d |", c[i]);
    // Cleanup   
    free(a); 
    free(b); 
    free(c);   
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c);   
 return 0;  
} 